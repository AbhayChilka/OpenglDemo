#include <stdio.h>

int main(void)
{
    //function declarations
    void PrintCUDADeviceProperties(void);

    //code
    PrintCUDADeviceProperties();
}

void PrintCUDADeviceProperties(void)
{
    //code
    printf("CUDA INFORMATION : \n");
    printf("===================================================\n");

    hipError_t ret_cuda_rt;
    int dev_count;
    ret_cuda_rt = hipGetDeviceCount(&dev_count);
    if (ret_cuda_rt != hipSuccess)
    {
        printf("CUDA Runtime API Error - hipGetDeviceCount \n", hipGetErrorString(ret_cuda_rt));
    }
    else if (dev_count == 0)
    {
        printf("There is no CUDA Supported Device On this System\n");
        return;
    }
    else
    {
        printf("Total Number of CUDA Supporting GPU Device\Devices on this System : %d\n", dev_count);
        for (int i = 0; i < dev_count; i++)
        {
            hipDeviceProp_t dev_prop;
            int driverVersion = 0, runtimeVersion = 0;

            ret_cuda_rt = hipGetDeviceProperties(&dev_prop, i);
            if (ret_cuda_rt != hipSuccess)
            {
                printf("%s in %s at line %d\n", hipGetErrorString (ret_cuda_rt), __FILE__, __LINE__);
                return;
            }
            printf("\n");
            hipDriverGetVersion(&driverVersion);
            hipRuntimeGetVersion(&runtimeVersion);
            printf("******CUDA DRIVER AND RUNTIME INFORMATION******\n");
            printf("==========================================================");
            printf("CUDA Driver Version               :%d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
            printf("CUDA Runtime Version              :%d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);
            printf("\n");
            printf("=========================================================");
            printf("******GPU DEVICE GENERAL INFORMATION******\n");
            printf("===========================================================");
            printf("GPU Device Number                 :%d\n", i);
            printf("GPU Device Name                   :%s\n", dev_prop.name);
            printf("GPU Device Compute Capability            :%d\n", dev_prop.major, dev_prop.minor);
            printf("GPU Device Clock Rate              :%d\n", dev_prop.clockRate);
            printf("GPU Device Type                    :");
            if (dev_prop.integrated)
                printf("Integrated (on-Board)\n");
            else
                printf("Discrete (card)\n");
            printf("\n");
            printf("****** GPU DEVICE MEMORY INFORMATION******\n");
            printf("==========================================================\n");
            printf("GPU Device Total Memory              :%.0fGB = %.0f MB = %llu Bytes\n", ((float)dev_prop.totalGlobalMem / 1048576.0f) / 1024.0f, (unsigned long  long) dev_prop.totalGlobalMem);
            printf("GPU Device Constant Memory                %lu Bytes\n", (unsigned long)dev_prop.totalConstMem);
            printf("GPU Device Shared Memory Per SMProcessor         :%lu\n", (unsigned long)dev_prop.sharedMemPerBlock);
            printf("\n");
            printf("******GPU DEVICE MULTIPROCESSOR INFORMATION******\n");
            printf("===========================================================\n");
            printf("GPU Device Number of SMProcessors                   :%d\n", dev_prop.multiProcessorCount);
            printf("GPU Device Number Of Registers per SMProcessor           :%d\n", dev_prop.regsPerBlock);
            printf("\n");
            printf("******GPU DEVICE THREAD INFORMATION******\n");
            printf("============================================================\n");
            printf("GPU Device Maximum Number Of Threads Per SMProcessor   %d\n", dev_prop.maxThreadsPerMultiProcessor);
            printf("GPU Device Maximum Number of Threads per block     %d\n", dev_prop.maxThreadsPerBlock);
            printf("GPU Device Threads in Warp            :%d\n", dev_prop.warpSize);
            printf("GPU Device Maximun Threads Dimensions        :(%d, %d, %d)\n", dev_prop.maxThreadsDim[0], dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);
            printf("GPU Device Maximum Grid Dimensions         :(%d, %d, %d)\n", dev_prop.maxGridSize[0], dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);
            printf("\n");
            printf("******GPU DEVICE DRIVER INFORMATION******\n");
            printf("=============================================\n");
            printf("GPU Device has ECC support \n", dev_prop.ECCEnabled ? "Enabled" : "Disabled");
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
        printf("GPU Device CUDA Driver Mode (TCC Or WDDM)  : %s\n", dev_prop.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDMM (Windows Display Driver Model)");
#endif 
    printf("***********************************************************************************************************************************************************************\n");
        }    
    }
}
