#include "hip/hip_runtime.h"
#include"SineWave.cu.h"

//cuda kernel for sinwave
__global__ void sinWaveKernel(float4 *pos, int width, int height, float time)
{
    //code
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = (float)x / (float)width;
	float v = (float)y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

    float frequency = 4.0f;
	float w = sinf(u * frequency + time) * cosf(v * frequency + time) * 0.5f;

    pos[y * width + x] =  make_float4(u, w, v, 1.0f);
}

//user define function to call cuda kernel
void launchCudaKernel(float4 *pos, int width, int heigth, float time)
{
    dim3 block(8, 8, 1);
    dim3 grid(width / block.x, heigth / block.y, 1);

    sinWaveKernel << <grid, block>> > (pos, width, heigth, time);
}