#include "hip/hip_runtime.h"
#include "SignWave.cu.h"

// Cuda kernel for signwave
__global__ void signWaveKernel(float4* pos, int width, int height, float time)
{
	// Code
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = (float)x / (float)width;
	float v = (float)y / (float)height;

	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	float frequency = 4.0f;
	float w = sinf(u * frequency + time) * cosf(v * frequency + time) * 0.5f;

	pos[y * width + x] = make_float4(u, w, v, 1.0f);
}

//User defined function to call Cuda Kernel
void launchCUDAKernel(float4* pos, int width, int height, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x,  height / block.y, 1);
	signWaveKernel << < grid,block >> > (pos, width, height, time);
}

